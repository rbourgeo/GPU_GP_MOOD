#include "hip/hip_runtime.h"
#include "Forward_Euler.h"
#include "SSP_RK2.h"
#include "SSP_RK3.h"
#include "bc.h"
#include "exact_sol.h"
#include "gp_stencil.h"
#include "init.h"
#include "output.h"
#include "parameters.h"
#include "recons_flux_computation.h"
#include "set_equal.h"
#include "state.h"

int main()
{

  /*time measurement variables*/
  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //Integration Parameters

  /* Numerical Mesh Configuration */
  double dx = Lx / double(lf);
  double dy = Ly / double(nf);

  double dtfinal;

  double ell_;

  /*init and computation of GP_stencil in the class*/
  if (ell_over_dx == 0)
  {
    ell_ = ell;
  }
  else
  {
    ell_ = dx*ell_over_dx;
  }

  GP_stencil GP(Mord,ell_);
  GP.init(dx, dy);

  int *index, *d_index;
  double *zT, *d_zT;

  zT = new double[4*nop*ngp]; //4 faces, 2*mord-1 points, ngp points per face
  index = new int[2*nop    ]; // x and y

  for (int k = 0; k <= nop-1; k++) {
    for (int r = 0; r <= ngp-1; r++) {

      zT[iL*(nop*ngp) + r*nop + k] = GP.zT_L[a2l(k, r)];
      zT[iT*(nop*ngp) + r*nop + k] = GP.zT_T[a2l(k, r)];
      zT[iR*(nop*ngp) + r*nop + k] = GP.zT_R[a2l(k, r)];
      zT[iB*(nop*ngp) + r*nop + k] = GP.zT_B[a2l(k, r)];

      index[dir_x*nop + k] = GP.index_x[k];
      index[dir_y*nop + k] = GP.index_y[k];

    }
  }


  GP._deallocate();
  /* Send the GP stencil data to the device */
  hipMalloc(&d_index, 2*nop*    sizeof(int   ));
  hipMalloc(&d_zT   , 4*nop*ngp*sizeof(double));

  hipMemcpy(d_zT,    zT   , 4*nop*ngp*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_index, index, 2*nop*    sizeof(int   ), hipMemcpyHostToDevice);


  /* Time stepping parameters */

  //double dt = CFL * pow(1.0f / (abs(ax) / dx + abs(ay) / dy),5./3);
  double dt = CFL *1.0f / (abs(ax) / dx + abs(ay) / dy);


  double t = 0.0f, tio = 0.5f;

  //Allocate Memory

  size_t total_size     =     ne * le * sizeof(double);
  size_t total_size_sol = 4 * ne * le * sizeof(double); /* 4 conserved quantities*/

  size_t size_x  = le  *       sizeof(double);
  size_t size_y  = ne  *       sizeof(double);
  size_t size_gw = ngp * ngp * sizeof(double);

  double *f,   *x,   *y,   *gauss_weight;
  double *d_f, *d_x, *d_y, *d_fout,      *d_fluxes_x, *d_fluxes_y, *d_f1, *d_f2, *d_gauss_weight;

  f = new double[4 * le * ne];
  x = new double[le         ];
  y = new double[ne         ];

  gauss_weight = new double[ngp * ngp];

  gauss_weight[a2l_ngp(0, 0)] = 1.;

  if (ngp >= 2) {
    gauss_weight[a2l_ngp(1, 0)] = 0.5;
    gauss_weight[a2l_ngp(1, 1)] = 0.5;
  }

  if (ngp >= 3) {
    gauss_weight[a2l_ngp(2, 0)] = 0.5 * 5. / 9;
    gauss_weight[a2l_ngp(2, 1)] = 0.5 * 8. / 9;
    gauss_weight[a2l_ngp(2, 2)] = 0.5 * 5. / 9;
  }

  //Send the gaussian point to the device
  hipMalloc(&d_gauss_weight, size_gw);
  hipMemcpy( d_gauss_weight, gauss_weight, size_gw, hipMemcpyHostToDevice);


  hipMalloc(&d_f       , total_size_sol);
  hipMalloc(&d_f1      , total_size_sol);
  hipMalloc(&d_f2      , total_size_sol);
  hipMalloc(&d_fout    , total_size_sol);
  hipMalloc(&d_fluxes_x, total_size_sol);
  hipMalloc(&d_fluxes_y, total_size_sol);
  hipMalloc(&d_x, size_x);
  hipMalloc(&d_y, size_y);

  //Apply Initial Condition
  initialize<<<dimGrid, dimBlock>>>(d_f, d_x, d_y, dx, dy);

  //Copy for IO operation
  hipMemcpy(x, d_x, size_x, hipMemcpyDeviceToHost);
  hipMemcpy(y, d_y, size_y, hipMemcpyDeviceToHost);

  /*====================== Perform Integration =======================*/
  std::string f2;
  int kk = 0;

  hipEventRecord(start, 0); // We only measure the computation time

  while (t < tmax) {

    dtfinal =  tmax-t;
    if (dt>dtfinal) {
      dt= dtfinal;
    }

    if (time_method == SSP_RK1) {

      Forward_Euler(d_f, d_fout,              d_fluxes_x, d_fluxes_y, dx, dy, dt, d_zT, d_index, d_gauss_weight);

    } else if (time_method == SSP_RK2) {
      SSP_RK2_(d_f, d_fout, d_f1,        d_fluxes_x, d_fluxes_y, dx, dy, dt, d_zT, d_index, d_gauss_weight);
    } else if (time_method == SSP_RK3) {
      SSP_RK3_(d_f, d_fout, d_f1, d_f2,  d_fluxes_x, d_fluxes_y, dx, dy, dt, d_zT, d_index, d_gauss_weight);
    }

    set_equal<<<dimGrid, dimBlock>>>(d_f, d_fout); //d_f = d_fout
    hipDeviceSynchronize();

    //std::cout<<t<<std::endl;
    t += dt;
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("Time to compute:  %3.1f ms \n", time); //faire mieux

  //Final output
  f2 = "output/final_sol.dat";
  hipMemcpy(f, d_f, total_size_sol, hipMemcpyDeviceToHost);

  std::cout << "Error = " << error_(f, x, y, dx, dy) << std::endl;
  io_fun(f2, x, y, f);

  //deallocate memory
  delete x;
  delete y;
  delete f;
  delete zT;
  delete index;
  hipFree(d_f);
  hipFree(d_f1);
  hipFree(d_f2);
  hipFree(d_gauss_weight);
  hipFree(d_fluxes_x);
  hipFree(d_fluxes_y);
  hipFree(d_fout);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_zT);
  hipFree(d_index);



  return 0;
}

//	if(fmod(t, tio) == 0.0f)
//	{
//	//IO function
//	f2 = "sol" + std::to_string(kk) + ".dat";
//	hipMemcpy(f,d_f, total_size, hipMemcpyDeviceToHost);
//	io_fun(f2, x, y, f);
//	kk++;
//	std::cout<< "output at t = "<<t<< std::endl;
//}
