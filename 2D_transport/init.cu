#include "hip/hip_runtime.h"
#include "exact_sol.h"
#include "init.h"
#include "parameters.h"

__global__ void initialize(double f[], double x[], double y[], const double dx, const double dy)
{
    int tidx = c2f(threadIdx.x + blockIdx.x * blockDim.x);
    int tidy = c2f(threadIdx.y + blockIdx.y * blockDim.y);

    double xt = (double(tidx) - 0.5f) * dx; //centers of cells
    double yt = (double(tidy) - 0.5f) * dy;

    if (tidx <= lf + ngc) {
        x[f2c(tidx)] = xt;
    }

    if (tidy <= nf + ngc) {
        y[f2c(tidy)] = yt;
    }

    if (tidx <= lf + ngc) {
        if (tidy <= nf + ngc) {
            //f[ij(tidy, tidx)] = exp(-100.0f * ((xt-0.5f) * (xt-0.5f) + (yt-0.5f) * (yt-0.5f)));
            f[ij(tidy, tidx)] = exact_soln(tidx, tidy, dx, dy, x, y);
        }
    }
}
