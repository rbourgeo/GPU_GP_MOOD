#include "hip/hip_runtime.h"
#include "Forward_Euler.h"
#include "SSP_RK2.h"
#include "SSP_RK3.h"
#include "bc.h"
#include "exact_sol.h"
#include "gp_stencil.h"
#include "init.h"
#include "output.h"
#include "parameters.h"
#include "recons_flux_computation.h"
#include "set_equal.h"
#include "state.h"

int main()
{

  /*time measurement variables*/
  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //Integration Parameters

  /* Numerical Mesh Configuration */
  double dx = Lx / double(lf);
  double dy = Ly / double(nf);

  double dtfinal;

  double ell_;

  /*init and computation of GP_stencil in the class*/
  if (ell_over_dx == 0) {
  ell_ = ell;}
  else{
    ell_ = dx*ell_over_dx;
  }

  GP_stencil GP(Mord,ell_);
  GP.init(dx, dy);

  int *index, *d_index;
  double *zT, *d_zT;

  zT = new double[4*nop*ngp]; //4 faces, 2*mord-1 points, ngp points per face
  index = new int[2*nop]; // x and y

  for (int k = 0; k <= nop-1; k++) {
    for (int r = 0; r <= ngp-1; r++) {

      zT[iL*(nop*ngp) + r*nop + k] = GP.zT_L[a2l(k, r)];
      zT[iT*(nop*ngp) + r*nop + k] = GP.zT_T[a2l(k, r)];
      zT[iR*(nop*ngp) + r*nop + k] = GP.zT_R[a2l(k, r)];
      zT[iB*(nop*ngp) + r*nop + k] = GP.zT_B[a2l(k, r)];

      index[dir_x*nop + k] = GP.index_x[k];
      index[dir_y*nop + k] = GP.index_y[k];

    }
  }

  for (int k = 0; k <= 2*Mord-2; k++) {
    std::cout<<"x "<< k<<" " <<index[dir_x*nop + k]<<std::endl;
    std::cout<<"y "<< k<<" " <<index[dir_y*nop + k]<<std::endl;

  }

  std::cout<<"L,r=1" << std::endl;

  for (int k = 0; k <= 2*5-2; k++) {
    std::cout<<zT[iL*nop*ngp + 0*nop+ k]<<std::endl;
  }

  std::cout<<"L,r=2" << std::endl;

  for (int k = 0; k <= 2*5-2; k++) {
    std::cout<<zT[iL*nop*ngp + 1*nop + k]<<std::endl;
  }



  std::cout<<"T,r=1" << std::endl;

  for (int k = 0; k <= 2*5-2; k++) {
    std::cout<<zT[iT*nop*ngp + 0*nop + k]<<std::endl;
  }

  std::cout<<"T,r=2" << std::endl;

  for (int k = 0; k <= 2*5-2; k++) {
    std::cout<<zT[iT*nop*ngp + 1*nop + k]<<std::endl;
  }


  std::cout<<"R,r=1" << std::endl;

  for (int k = 0; k <= 2*5-2; k++) {
    std::cout<<zT[iR*nop*ngp + 0*nop + k]<<std::endl;
  }

  std::cout<<"R,r=2" << std::endl;

  for (int k = 0; k <= 2*5-2; k++) {
    std::cout<<zT[iR*nop*ngp + 1*nop + k]<<std::endl;
  }


  std::cout<<"Br=1" << std::endl;

  for (int k = 0; k <= 2*5-2; k++) {
    std::cout<<zT[iB*nop*ngp + 0*nop + k]<<std::endl;
  }

  std::cout<<"B,r=2" << std::endl;

  for (int k = 0; k <= 2*5-2; k++) {
    std::cout<<zT[iB*nop*ngp + 1*nop + k]<<std::endl;
  }





  GP._deallocate();
  /* Send the GP stencil data to the device */
  hipMalloc(&d_index, 2*nop*sizeof(int));
  hipMalloc(&d_zT, 4*nop*ngp*sizeof(double));

  hipMemcpy(d_zT, zT, 4*nop*ngp*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_index, index, 2*nop*sizeof(int), hipMemcpyHostToDevice);


  /* Time stepping parameters */

  double dt = CFL * pow(1.0f / (abs(ax) / dx + abs(ay) / dy),5./3);

  double t = 0.0f, tio = 0.5f;

  //Allocate Memory

  size_t total_size = ne * le * sizeof(double);
  size_t size_x = le * sizeof(double);
  size_t size_y = ne * sizeof(double);
  size_t size_gw = ngp * ngp * sizeof(double);

  double *f, *x, *y, *gauss_weight;
  double *d_f, *d_x, *d_y, *d_fout, *d_fluxes_x, *d_fluxes_y, *d_f1, *d_f2, *d_gauss_weight;

  f = new double[le * ne];
  x = new double[le];
  y = new double[ne];

  gauss_weight = new double[ngp * ngp];

  gauss_weight[a2l_ngp(0, 0)] = 1.;

  if (ngp >= 2) {
    gauss_weight[a2l_ngp(1, 0)] = 0.5;
    gauss_weight[a2l_ngp(1, 1)] = 0.5;
  }

  if (ngp >= 3) {
    gauss_weight[a2l_ngp(2, 0)] = 0.5 * 5. / 9;
    gauss_weight[a2l_ngp(2, 1)] = 0.5 * 8. / 9;
    gauss_weight[a2l_ngp(2, 2)] = 0.5 * 5. / 9;
  }

  //Send the gaussian point to the device
  hipMalloc(&d_gauss_weight, size_gw);
  hipMemcpy(d_gauss_weight, gauss_weight, size_gw, hipMemcpyHostToDevice);


  hipMalloc(&d_f, total_size);
  hipMalloc(&d_f1, total_size);
  hipMalloc(&d_f2, total_size);
  hipMalloc(&d_fout, total_size);
  hipMalloc(&d_fluxes_x, total_size);
  hipMalloc(&d_fluxes_y, total_size);
  hipMalloc(&d_x, size_x);
  hipMalloc(&d_y, size_y);

  std::cout << "----------TEST-----------" << std::endl;
  std::cout << ij(-ngc + le, -ngc + le) << " " << (int(le) * int(ne) - 1) << std::endl;
  std::cout << ij(1 - ngc, 1 - ngc) << " " << 0 << std::endl;
  std::cout << "----------TEST-----------" << std::endl;

  //Apply Initial Condition
  initialize<<<dimGrid, dimBlock>>>(d_f, d_x, d_y, dx, dy);

  //Copy for IO operation
  hipMemcpy(x, d_x, size_x, hipMemcpyDeviceToHost);
  hipMemcpy(y, d_y, size_y, hipMemcpyDeviceToHost);

  /*====================== Perform Integration =======================*/
  std::string f2;
  int kk = 0;

  hipEventRecord(start, 0); // We only measure the computation time

  while (t < tmax) {

    dtfinal =  tmax-t;
    if (dt>dtfinal) {
      dt= dtfinal;
    }

    if (time_method == SSP_RK1) {
      Forward_Euler(d_f, d_fout,              d_fluxes_x, d_fluxes_y, dx, dy, dt, d_zT, d_index, d_gauss_weight);
    } else if (time_method == SSP_RK2) {
           SSP_RK2_(d_f, d_fout, d_f1,        d_fluxes_x, d_fluxes_y, dx, dy, dt, d_zT, d_index, d_gauss_weight);
    } else if (time_method == SSP_RK3) {
           SSP_RK3_(d_f, d_fout, d_f1, d_f2,  d_fluxes_x, d_fluxes_y, dx, dy, dt, d_zT, d_index, d_gauss_weight);
    }

    set_equal<<<dimGrid, dimBlock>>>(d_f, d_fout); //d_f = d_fout
    hipDeviceSynchronize();

    //std::cout<<t<<std::endl;
    t += dt;
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("Time to compute:  %3.1f ms \n", time); //faire mieux

  //Final output
  f2 = "output/final_sol.dat";
  hipMemcpy(f, d_f, total_size, hipMemcpyDeviceToHost);

  std::cout << "Error = " << error_(f, x, y, dx, dy) << std::endl;
  io_fun(f2, x, y, f);

  //deallocate memory
  delete x;
  delete y;
  delete f;
  delete zT;
  delete index;
  hipFree(d_f);
  hipFree(d_f1);
  hipFree(d_f2);
  hipFree(d_gauss_weight);
  hipFree(d_fluxes_x);
  hipFree(d_fluxes_y);
  hipFree(d_fout);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_zT);
  hipFree(d_index);



  return 0;
}

//	if(fmod(t, tio) == 0.0f)
//	{
//	//IO function
//	f2 = "sol" + std::to_string(kk) + ".dat";
//	hipMemcpy(f,d_f, total_size, hipMemcpyDeviceToHost);
//	io_fun(f2, x, y, f);
//	kk++;
//	std::cout<< "output at t = "<<t<< std::endl;
//}
